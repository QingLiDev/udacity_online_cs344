#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
//#include <thrust/host_vector.h>
#include "timer.h"

/* Red Eye Removal
   ===============

   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
   For example [0 0 1 1 0 0 1]
   ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
   output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

*/

using namespace std;

__global__ void calcHisto(unsigned int *d_histo, unsigned int *d_in, unsigned int mask, int i, size_t numElems) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= (int)numElems) return;
  unsigned int bin = (d_in[gid] & mask) >> i;
  atomicAdd(&d_histo[bin], 1);
}

__global__ void calcOneBefore(unsigned int *d_in, unsigned int *d_oneBefore, unsigned int mask, int i, size_t numElems) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= (int)numElems) return;
  if (((d_in[gid] & mask) >> i) == 1) {
    for (size_t k = gid+1; k < numElems; k++)
      atomicAdd(&d_oneBefore[k], 1);
  }
}

__global__ void movePos(unsigned int *d_valSrc, unsigned int *d_posSrc,
                        unsigned int *d_valDst, unsigned int *d_posDst,
                        unsigned int *d_scan, unsigned int *d_oneBefore,
                        unsigned int mask, int i, size_t numElems) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= (int)numElems) return;
  unsigned int bin = (d_valSrc[gid] & mask) >> i;
  if (bin == 0) {
    d_valDst[d_scan[bin] + gid - d_oneBefore[gid]] = d_valSrc[gid];
    d_posDst[d_scan[bin] + gid - d_oneBefore[gid]] = d_posSrc[gid];
  } else {
    d_valDst[d_scan[bin] + d_oneBefore[gid]] = d_valSrc[gid];
    d_posDst[d_scan[bin] + d_oneBefore[gid]] = d_posSrc[gid];
  }
}

__global__ void copy(unsigned int *d_inVal, unsigned int *d_inPos,
                     unsigned int * d_outVal, unsigned int *d_outPos,
                     size_t numElems) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= (int)numElems) return;
  d_outVal[gid] = d_inVal[gid];
  d_outPos[gid] = d_inPos[gid];
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
  //TODO
  //PUT YOUR SORT HERE
  std::cout << "numElems = " << numElems << std::endl;

  const int nBins = 2;
  const dim3 blkDim(512, 1, 1);
  const dim3 grdDim(ceil(numElems/(double)blkDim.x), 1, 1);
  cout << "blkDim.x=" << blkDim.x << "\tgrdDim.x=" << grdDim.x << endl;

  //unsigned int *h_oneBefore = new unsigned int[numElems];
  unsigned int *d_histo, *d_scan, *d_oneBefore;
  size_t bin_size = nBins * sizeof(unsigned int);
  size_t ele_size = numElems * sizeof(unsigned int);
  checkCudaErrors(hipMalloc(&d_histo, bin_size));
  checkCudaErrors(hipMalloc(&d_scan, bin_size));
  checkCudaErrors(hipMalloc(&d_oneBefore, ele_size));

  unsigned int *d_valSrc = d_inputVals;
  unsigned int *d_posSrc = d_inputPos;
  unsigned int *d_valDst = d_outputVals;
  unsigned int *d_posDst = d_outputPos;

  for (size_t i = 0; i < 8 * sizeof(unsigned int); i++) {
    unsigned int mask = 1 << i;
    checkCudaErrors(hipMemset(d_histo, 0, bin_size));
    checkCudaErrors(hipMemset(d_scan, 0, bin_size));
    checkCudaErrors(hipMemset(d_oneBefore, 0, ele_size));

    // calculate histogram
    calcHisto<<<grdDim, blkDim>>>(d_histo, d_valSrc, mask, i, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // sum scan -- since there are only two bins, it can be done by memcpy
    hipMemcpy(&d_scan[1], d_histo, sizeof(unsigned int), hipMemcpyDeviceToDevice);

    // count d_oneBefore
    // GpuTimer timer;
    // timer.Start();
    calcOneBefore<<<grdDim, blkDim>>>(d_valSrc, d_oneBefore, mask, i, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    // timer.Stop();
    // cout << timer.Elapsed() << endl;

    // move to the right position
    movePos<<<grdDim, blkDim>>>(d_valSrc, d_posSrc, d_valDst, d_posDst, d_scan, d_oneBefore, mask, i, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    std::swap(d_valDst, d_valSrc);
    std::swap(d_posDst, d_posSrc);
  }

  copy<<<grdDim, blkDim>>>(d_inputVals, d_inputPos, d_outputVals, d_outputPos, numElems);

  checkCudaErrors(hipFree(d_histo));
  checkCudaErrors(hipFree(d_scan));
  checkCudaErrors(hipFree(d_oneBefore));
}
